#include "hip/hip_runtime.h"
#include "GpuAcceleratedReliefFBucketSortProcessor.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utils/Timer.h"
#include "GpuAcceleratedReliefFCommonMethods.h"

using namespace std;

GpuAcceleratedReliefFBucketSortProcessor::GpuAcceleratedReliefFBucketSortProcessor(int kNearest){
	parallelizationType = PARALLELIZE_ON_STAGES;
	kNearestInstance = kNearest;
}

int GpuAcceleratedReliefFBucketSortProcessor::getKNearest(){
	return kNearestInstance;
}

Result* GpuAcceleratedReliefFBucketSortProcessor::parallelizeCalculationOnStages(int numOfSamples, int numOfFeatures, char* sampleFeatureMatrix, int* packedSampleFeatureMatrix, bool* featureMask, char* labels){
	
	if(isDebugEnabled()){
		cout<<"numOfSamples="<<numOfSamples<<", numOfFeatures="<<numOfFeatures<<endl;
	}
	
	Timer processing("Processing");
	processing.start();
	
	int kNearest = getKNearest();
	
	bool* d_featureMask;	
	int* d_packedSampleFeatureMatrix;	
	int* d_hitDistanceBuckets;
	int* d_missDistanceBuckets;
	char* d_labels;
		
	//int intsPerInstance = numOfFeatures / 16 + (numOfFeatures % 16 == 0? 0 : 1);
	int intsPerInstance = (int)ceil((float)numOfFeatures / 16);	
	
	hipMalloc(&d_featureMask, numOfFeatures*sizeof(bool));
	hipMemcpy(d_featureMask, featureMask, numOfFeatures*sizeof(bool),hipMemcpyHostToDevice);
	getMemoryInfo("after featureMask hipMalloc");
	
	hipMalloc(&d_packedSampleFeatureMatrix, intsPerInstance * numOfSamples*sizeof(int));	
	hipMemcpy(d_packedSampleFeatureMatrix, packedSampleFeatureMatrix, intsPerInstance * numOfSamples*sizeof(int),hipMemcpyHostToDevice);	
	getMemoryInfo("after packedSampleFeatureMatrix hipMalloc");
	
	hipMalloc(&d_labels, numOfSamples*sizeof(char));
	hipMemcpy(d_labels, labels, numOfSamples*sizeof(char),hipMemcpyHostToDevice);
	getMemoryInfo("after labels hipMalloc");	
	
	
	hipMalloc(&d_hitDistanceBuckets, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	hipMemset(d_hitDistanceBuckets, 0, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	getMemoryInfo("after hitDistanceBuckets hipMalloc");
	
	hipMalloc(&d_missDistanceBuckets, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	hipMemset(d_missDistanceBuckets, 0, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	getMemoryInfo("after missDistanceBuckets hipMalloc");
	

	int* d_kNearestHit;
	int* d_kNearestMiss;
	hipMalloc(&d_kNearestHit, kNearest * numOfSamples*sizeof(int));
	getMemoryInfo("after kNearestHit hipMalloc");
	hipMalloc(&d_kNearestMiss, kNearest * numOfSamples*sizeof(int));
	getMemoryInfo("after kNearestMiss hipMalloc");
		
	float* finalWeight = (float*)calloc(numOfFeatures,sizeof(float));
	float* d_weight;
	float* d_finalWeight;
	
	hipMalloc(&d_weight, numOfSamples*numOfFeatures*sizeof(float));
	hipMemset(d_weight, 0, numOfSamples*numOfFeatures*sizeof(float));
	getMemoryInfo("after weight hipMalloc");
	
	
	hipMalloc(&d_finalWeight, numOfFeatures*sizeof(float));	
	hipMemset(d_finalWeight, 0, numOfFeatures*sizeof(float));
	getMemoryInfo("after finalWeight hipMalloc");
		
	int grid2d = (int)ceil(pow(numOfSamples,1/2.));
	int threadSize = getNumberOfThreadsPerBlock();
		
	int samplePerThread = (int)ceil(((float)numOfSamples)/threadSize);	
	
	if(isDebugEnabled()){
		cout<<"grid size="<<grid2d<<"x"<<grid2d<<endl;
		cout<<"thread size="<<threadSize<<endl;
		cout<<"samplePerThread="<<samplePerThread<<endl;
	}
	
	dim3 gridSize(grid2d,grid2d);
	
	if(isDebugEnabled()){
		cout<<"generate distance buckets"<<endl;
	}
	gpu_generateDisatanceBuckets<<<gridSize, threadSize>>>(	
		kNearest,
		samplePerThread,
		numOfSamples,			
		numOfFeatures,		
		intsPerInstance,
		d_labels,		
		d_packedSampleFeatureMatrix,		
		d_hitDistanceBuckets,
		d_missDistanceBuckets
		);
	hipDeviceSynchronize();		
	
	if(this->isDebugEnabled()){		
		cout<<"hipPeekAtLastError:"<<hipGetErrorString(hipPeekAtLastError())<<endl;
	}
	
	if(isDebugEnabled()){
		cout<<"weight features"<<endl;
	}	
	
	gpu_weightFeatures<<<gridSize,1>>>(
		kNearest,
		numOfFeatures,
		numOfSamples,
		intsPerInstance,
		d_kNearestHit,
		d_kNearestMiss,
		d_hitDistanceBuckets,
		d_missDistanceBuckets,
		d_featureMask,
		d_packedSampleFeatureMatrix,
		d_weight,
		d_finalWeight
	);
			
	hipDeviceSynchronize();
	
	if(this->isDebugEnabled()){		
		cout<<"hipPeekAtLastError:"<<hipGetErrorString(hipPeekAtLastError())<<endl;
	}
	
	hipMemcpy(finalWeight, d_finalWeight, numOfFeatures*sizeof(float), hipMemcpyDeviceToHost);
			
	if(isDebugEnabled()){
		cout<<"generate result"<<endl;
	}
	Result* result = new Result;
	result->scores = new double[numOfFeatures];
	int divider = numOfSamples * kNearest;
	for(int i=0;i<numOfFeatures;i++){
		result->scores[i] = finalWeight[i]/divider;
	}
	result->success = true;	
		
	free(finalWeight);
	
	hipFree(d_packedSampleFeatureMatrix);	
	hipFree(d_labels);
	hipFree(d_hitDistanceBuckets);
	hipFree(d_missDistanceBuckets);
	hipFree(d_featureMask);	
	hipFree(d_labels);
	hipFree(d_kNearestHit);
	hipFree(d_kNearestMiss);
	hipFree(d_weight);
	hipFree(d_finalWeight);
	
	processing.stop();
	result->startTime=processing.getStartTime();
	result->endTime=processing.getStopTime();	
	
	return result;
}

