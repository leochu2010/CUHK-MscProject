#include "hip/hip_runtime.h"
#include "GpuAcceleratedReliefFBucketSortProcessor.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utils/Timer.h"

using namespace std;

GpuAcceleratedReliefFBucketSortProcessor::GpuAcceleratedReliefFBucketSortProcessor(int kNearest){
	parallelizationType = PARALLELIZE_ON_STAGES;
	kNearestInstance = kNearest;
}

int GpuAcceleratedReliefFBucketSortProcessor::getKNearest(){
	return kNearestInstance;
}

__device__ void gpu_pushSampleIdIntoBucket(int sample1Id, int sample2Id, int numOfFeatures, int distance, int kNearest, int* d_distanceBuckets){
		
		int sampleBucketNum = sample1Id * numOfFeatures * (kNearest+1) + (kNearest+1) * distance;
		if(d_distanceBuckets[sampleBucketNum] < kNearest){				
		
			int sampleBucketIdx = atomicAdd(&d_distanceBuckets[sampleBucketNum], 1) + 1;			
			if (sampleBucketIdx > kNearest){
				atomicSub(&d_distanceBuckets[sampleBucketNum], 1);
			}else{
				atomicAdd(&d_distanceBuckets[sampleBucketNum + sampleBucketIdx], sample2Id);
			}
		
		}	
}

__global__ void gpu_generateDisatanceBuckets(
		int kNearest,
		int samplePerThread,
		int numOfSamples,			
		int numOfFeatures,		
		int intsPerInstance,
		char* d_labels,		
		int* d_packedSampleFeatureMatrix,		
		int* d_hitDistanceBuckets,
		int* d_missDistanceBuckets		
	){
		
	int sample1Id = gridDim.x * blockIdx.x + blockIdx.y;	

	if(sample1Id >= numOfSamples){
		return;
	}

	/*
	if(threadIdx.x == 0){
		printf("sample1=%d, gridDim.x=%d * blockIdx.x=%d + blockIdx.y=%d\n", sample1Id,gridDim.x,blockIdx.x,blockIdx.y);
	}
	*/
	
	for(int i = 0; i < samplePerThread; i++){
		
		int sample2Id = threadIdx.x * samplePerThread + i;
		
		if(sample2Id <= sample1Id){
			continue;
		}
		
		if(sample2Id >= numOfSamples){
			break;
		}
					
		int distance = 0;
		for(int k = 0; k < intsPerInstance; k++){
			int first = d_packedSampleFeatureMatrix[sample1Id * intsPerInstance + k];
			int second = d_packedSampleFeatureMatrix[sample2Id * intsPerInstance + k];
			/*
			if(threadIdx.x == 0 && k==0){
				printf("first=%d, second=%d\n", first, second);
			}*/
			int ret = first ^ second;
			for(int l = 0; l < 32; l += 2){
				int diff = (ret >> l) & 3;
				if(diff != 0) distance++; 
			}
		}
		
		if(d_labels[sample1Id] == d_labels[sample2Id]){			
			gpu_pushSampleIdIntoBucket(sample1Id, sample2Id, numOfFeatures, distance, kNearest, d_hitDistanceBuckets);
			gpu_pushSampleIdIntoBucket(sample2Id, sample1Id, numOfFeatures, distance, kNearest, d_hitDistanceBuckets);
		}else{			
			gpu_pushSampleIdIntoBucket(sample1Id, sample2Id, numOfFeatures, distance, kNearest, d_missDistanceBuckets);
			gpu_pushSampleIdIntoBucket(sample2Id, sample1Id, numOfFeatures, distance, kNearest, d_missDistanceBuckets);
		}
	}
}

__device__ void gpu_findKNearestFromBuckets(int numOfFeatures, int sampleId, int* d_distanceBuckets, int* d_kNearestSampleId, int kNearest){
	int numOfSamples = 0;	
	for(int distance=0; distance<numOfFeatures; distance++){	
			
		int bucket = sampleId * numOfFeatures * (kNearest+1) + (kNearest+1) * distance;
		
		/*
		if(sampleId == 0){			
			printf("d_distanceBuckets[%d]=%d, %d, %d, %d, %d, %d \n",distance, d_distanceBuckets[bucket], 
			d_distanceBuckets[bucket + 1],
			d_distanceBuckets[bucket + 2],
			d_distanceBuckets[bucket + 3],
			d_distanceBuckets[bucket + 4],
			d_distanceBuckets[bucket + 5]);
		}*/
		
		for(int i=0; i<d_distanceBuckets[bucket];i++){
			int nearSampleId = d_distanceBuckets[bucket + i+1];
			d_kNearestSampleId[sampleId * kNearest + numOfSamples] = nearSampleId;
			numOfSamples += 1;
			/*
			if(sampleId == 0){
				printf("sampleId:%d\n",nearSampleId);
			}*/
			if(numOfSamples == kNearest){
				return;
			}
		}
	}
}

__global__ void gpu_weightFeaturesFromBuckets(
		int kNearest,
		int numOfFeatures,
		int numOfSamples,
		int intsPerInstance,
		int* d_kNearestHit,
		int* d_kNearestMiss,
		int* d_hitDistanceBuckets,
		int* d_missDistanceBuckets,
		bool* d_featureMask,
		int* d_packedSampleFeatureMatrix,
		float* d_weight,
		float* d_finalWeight
	){
		
	int sampleId = gridDim.x * blockIdx.x + blockIdx.y;
	
	//printf("sample=%d, gridDim.x=%d * blockIdx.x=%d + blockIdx.y=%d\n", sampleId,gridDim.x,blockIdx.x,blockIdx.y);	
	
	
	if(sampleId >= numOfSamples){
		return;
	}
	
	gpu_findKNearestFromBuckets(numOfFeatures, sampleId, d_hitDistanceBuckets, d_kNearestHit, kNearest);
	gpu_findKNearestFromBuckets(numOfFeatures, sampleId, d_missDistanceBuckets, d_kNearestMiss, kNearest);
	
	/*
	if(sampleId == 999){
		for(int i=0;i<kNearest;i++){
			printf("k=%d, hitSampleId=%d, missSampleId=%d\n", i, d_kNearestHit[sampleId * kNearest+i], d_kNearestMiss[sampleId * kNearest+i]);
		}
	}
	*/	
	
	for(int k=0; k<kNearest; k++){
		int hitSampleId = d_kNearestHit[sampleId * kNearest + k];
		int missSampleId = d_kNearestMiss[sampleId * kNearest + k];
		
		for(int i=0;i<intsPerInstance;i++){
			int instanceInt = d_packedSampleFeatureMatrix[sampleId * intsPerInstance + i];
			int hitInt = d_packedSampleFeatureMatrix[hitSampleId * intsPerInstance + i];
			int missInt = d_packedSampleFeatureMatrix[missSampleId * intsPerInstance + i];

			for(int offset = 0; offset < 16; offset++)
			{
				int attributeIdx = i * 16 + offset;
				if(d_featureMask[attributeIdx] != true){
					continue;
				}
				
				if(attributeIdx < numOfFeatures)
				{
					int deltaHit = ((instanceInt >> offset * 2) & 0x3) == ((hitInt >> offset * 2) & 0x3)? 0 : 1;
					int deltaMiss = ((instanceInt >> offset * 2) & 0x3) == ((missInt >> offset * 2) & 0x3)? 0 : 1;
					float score = deltaMiss - deltaHit;
					d_weight[sampleId * numOfFeatures + attributeIdx] += score;					
				}
			}
		}
	}

/*		
		for(int j=0; j<numOfFeatures; j++){
			
			if(d_featureMask[j] != true){
				continue;
			}
			
			char feature = d_sampleFeatureMatrix[sampleId * numOfFeatures + j];
			char hitFeature = d_sampleFeatureMatrix[hitSampleId * numOfFeatures + j];
			char missFeature = d_sampleFeatureMatrix[missSampleId * numOfFeatures + j];
			
			if (feature != hitFeature){
				d_weight[sampleId * numOfFeatures + j] -= 1;
			}
			
			if (feature != missFeature){
				d_weight[sampleId * numOfFeatures + j] += 1;
			}
		}
*/
	
	for(int i=0; i<numOfFeatures; i++){						
		atomicAdd(&d_finalWeight[i],d_weight[sampleId * numOfFeatures + i]);
	}
	
}

Result* GpuAcceleratedReliefFBucketSortProcessor::parallelizeCalculationOnStages(int numOfSamples, int numOfFeatures, char* sampleFeatureMatrix, int* packedSampleFeatureMatrix, bool* featureMask, char* labels){
	
	if(isDebugEnabled()){
		cout<<"numOfSamples="<<numOfSamples<<", numOfFeatures="<<numOfFeatures<<endl;
	}
	
	Timer processing("Processing");
	processing.start();
	
	int kNearest = getKNearest();
	
	bool* d_featureMask;	
	int* d_packedSampleFeatureMatrix;	
	int* d_hitDistanceBuckets;
	int* d_missDistanceBuckets;
	char* d_labels;
		
	//int intsPerInstance = numOfFeatures / 16 + (numOfFeatures % 16 == 0? 0 : 1);
	int intsPerInstance = (int)ceil((float)numOfFeatures / 16);	
	
	hipMalloc(&d_featureMask, numOfFeatures*sizeof(bool));
	hipMemcpy(d_featureMask, featureMask, numOfFeatures*sizeof(bool),hipMemcpyHostToDevice);
	getMemoryInfo("after featureMask hipMalloc");
	
	hipMalloc(&d_packedSampleFeatureMatrix, intsPerInstance * numOfSamples*sizeof(int));	
	hipMemcpy(d_packedSampleFeatureMatrix, packedSampleFeatureMatrix, intsPerInstance * numOfSamples*sizeof(int),hipMemcpyHostToDevice);	
	getMemoryInfo("after packedSampleFeatureMatrix hipMalloc");
	
	hipMalloc(&d_labels, numOfSamples*sizeof(char));
	hipMemcpy(d_labels, labels, numOfSamples*sizeof(char),hipMemcpyHostToDevice);
	getMemoryInfo("after labels hipMalloc");	
	
	
	hipMalloc(&d_hitDistanceBuckets, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	hipMemset(d_hitDistanceBuckets, 0, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	getMemoryInfo("after hitDistanceBuckets hipMalloc");
	
	hipMalloc(&d_missDistanceBuckets, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	hipMemset(d_missDistanceBuckets, 0, numOfSamples * numOfFeatures * (kNearest+1) * sizeof(int));
	getMemoryInfo("after missDistanceBuckets hipMalloc");
	

	int* d_kNearestHit;
	int* d_kNearestMiss;
	hipMalloc(&d_kNearestHit, kNearest * numOfSamples*sizeof(int));
	getMemoryInfo("after kNearestHit hipMalloc");
	hipMalloc(&d_kNearestMiss, kNearest * numOfSamples*sizeof(int));
	getMemoryInfo("after kNearestMiss hipMalloc");
		
	float* finalWeight = (float*)calloc(numOfFeatures,sizeof(float));
	float* d_weight;
	float* d_finalWeight;
	
	hipMalloc(&d_weight, numOfSamples*numOfFeatures*sizeof(float));
	hipMemset(d_weight, 0, numOfSamples*numOfFeatures*sizeof(float));
	getMemoryInfo("after weight hipMalloc");
	
	
	hipMalloc(&d_finalWeight, numOfFeatures*sizeof(float));	
	hipMemset(d_finalWeight, 0, numOfFeatures*sizeof(float));
	getMemoryInfo("after finalWeight hipMalloc");
		
	int grid2d = (int)ceil(pow(numOfSamples,1/2.));
	int threadSize = getNumberOfThreadsPerBlock();
		
	int samplePerThread = (int)ceil(((float)numOfSamples)/threadSize);	
	
	if(isDebugEnabled()){
		cout<<"grid size="<<grid2d<<"x"<<grid2d<<endl;
		cout<<"thread size="<<threadSize<<endl;
		cout<<"samplePerThread="<<samplePerThread<<endl;
	}
	
	dim3 gridSize(grid2d,grid2d);
	
	if(isDebugEnabled()){
		cout<<"generate distance buckets"<<endl;
	}
	gpu_generateDisatanceBuckets<<<gridSize, threadSize>>>(	
		kNearest,
		samplePerThread,
		numOfSamples,			
		numOfFeatures,		
		intsPerInstance,
		d_labels,		
		d_packedSampleFeatureMatrix,		
		d_hitDistanceBuckets,
		d_missDistanceBuckets
		);
	hipDeviceSynchronize();		
	
	if(this->isDebugEnabled()){		
		cout<<"hipPeekAtLastError:"<<hipGetErrorString(hipPeekAtLastError())<<endl;
	}
	
	if(isDebugEnabled()){
		cout<<"weight features"<<endl;
	}	
	
	gpu_weightFeaturesFromBuckets<<<gridSize,1>>>(
		kNearest,
		numOfFeatures,
		numOfSamples,
		intsPerInstance,
		d_kNearestHit,
		d_kNearestMiss,
		d_hitDistanceBuckets,
		d_missDistanceBuckets,
		d_featureMask,
		d_packedSampleFeatureMatrix,
		d_weight,
		d_finalWeight
	);
			
	hipDeviceSynchronize();
	
	if(this->isDebugEnabled()){		
		cout<<"hipPeekAtLastError:"<<hipGetErrorString(hipPeekAtLastError())<<endl;
	}
	
	hipMemcpy(finalWeight, d_finalWeight, numOfFeatures*sizeof(float), hipMemcpyDeviceToHost);
			
	if(isDebugEnabled()){
		cout<<"generate result"<<endl;
	}
	Result* result = new Result;
	result->scores = new double[numOfFeatures];
	int divisor = numOfSamples * kNearest;
	for(int i=0;i<numOfFeatures;i++){
		result->scores[i] = finalWeight[i]/divisor;
	}
	result->success = true;	
		
	free(finalWeight);
	
	hipFree(d_packedSampleFeatureMatrix);	
	hipFree(d_labels);
	hipFree(d_hitDistanceBuckets);
	hipFree(d_missDistanceBuckets);
	hipFree(d_featureMask);	
	hipFree(d_labels);
	hipFree(d_kNearestHit);
	hipFree(d_kNearestMiss);
	hipFree(d_weight);
	hipFree(d_finalWeight);
	
	processing.stop();
	result->startTime=processing.getStartTime();
	result->endTime=processing.getStopTime();	
	
	return result;
}

