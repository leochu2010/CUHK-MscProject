#include "GpuAcceleratedProcessor.h"
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

GpuAcceleratedProcessor::GpuAcceleratedProcessor(){
	this->numberOfThreadsPerBlock = 0;
	this->numberOfDevice = 0;
	this->activated = false;	
}

void GpuAcceleratedProcessor::setNumberOfThreadsPerBlock(int numberOfThreadsPerBlock)
{
		this->numberOfThreadsPerBlock = numberOfThreadsPerBlock;
}

int GpuAcceleratedProcessor::getNumberOfThreadsPerBlock()
{
		if (this->numberOfThreadsPerBlock >0){
			return numberOfThreadsPerBlock;
		}else{
			return 1024;
		}
}

void GpuAcceleratedProcessor::setNumberOfDevice(int numberOfDevice){
	this->numberOfDevice = numberOfDevice;
}

int GpuAcceleratedProcessor::getNumberOfDevice(){
	
	if (this->numberOfDevice >0){
		return numberOfDevice;
	}else{
		int deviceCount = 0;
		hipGetDeviceCount(&deviceCount);
		return deviceCount;
	}
}

int GpuAcceleratedProcessor::getNumberOfFeatureSizeTimesSampleSize2dArrays(int numOfFeatures){
	return this->getNumberOfDevice();
}

Result* GpuAcceleratedProcessor::calculate(int numOfSamples, int numOfFeatures, char* sampleTimesFeature, bool* featureMask, char* labels){
	return new Result;
}

Result* GpuAcceleratedProcessor::calculate(int numOfFeatures, 
		char** label0ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel0Samples,
		char** label1ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel1Samples, 
		bool* featureMask){
		
		/*
		Step 1:
			for each device
				new thread for create stream
		
		Step 2:
			create results array
		
		Step 3:
			wait for all threads join 
		
		Step 4:
			for each device	
				new thread for asyn calculation
					pass data, devId, stream for asyn calculate
					pass resultArray for passing result back
		
		Step 5:
			wait for all threads join
		
		Step 6:
			return results;
		*/
		
		
		/*
			don't forget try stream features in warp size in child classes
		*/
			
	return new Result;
	
}

