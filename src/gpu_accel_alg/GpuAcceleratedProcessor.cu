#include "GpuAcceleratedProcessor.h"
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <string>
#include "threadpool/ThreadPool.h"
#include "utils/Timer.h"

using namespace std;

GpuAcceleratedProcessor::GpuAcceleratedProcessor(){
	this->numberOfThreadsPerBlock = 0;
	this->numberOfDevice = 0;
	this->numberOfStreamsPerDevice = 1;
	this->threadPoolEnabled = false;
}

void GpuAcceleratedProcessor::enableThreadPool(){
	this->threadPoolEnabled = true;
}

void GpuAcceleratedProcessor::setNumberOfThreadsPerBlock(int numberOfThreadsPerBlock)
{
		this->numberOfThreadsPerBlock = numberOfThreadsPerBlock;
}

int GpuAcceleratedProcessor::getNumberOfThreadsPerBlock()
{
		if (this->numberOfThreadsPerBlock >0){
			return numberOfThreadsPerBlock;
		}else{
			return 1024;
		}
}

void GpuAcceleratedProcessor::setNumberOfDevice(int numberOfDevice){
	this->numberOfDevice = numberOfDevice;
}

int GpuAcceleratedProcessor::getNumberOfDevice(){
	
	if (this->numberOfDevice >0){
		return numberOfDevice;
	}else{
		int deviceCount = 0;
		hipGetDeviceCount(&deviceCount);
		return deviceCount;
	}
}

void GpuAcceleratedProcessor::setNumberOfStreamsPerDevice(int numberOfStreamsPerDevice)
{
	this->numberOfStreamsPerDevice = numberOfStreamsPerDevice;
}

int GpuAcceleratedProcessor::getNumberOfStreamsPerDevice()
{
	return this->numberOfStreamsPerDevice;
}

struct CreateStreamArgs{
	hipStream_t* stream;
	hipError_t* streamResult;
	int numberOfStreamsPerDevice;
	int dev;
};

//create stream
void createStream(void* arg) {	
	CreateStreamArgs* createStreamArgs = (CreateStreamArgs*) arg;
	
	hipStream_t* stream = createStreamArgs->stream;
	hipError_t* streamResult = createStreamArgs->streamResult;
	int dev = createStreamArgs->dev;
	int numberOfStreamsPerDevice = createStreamArgs->numberOfStreamsPerDevice;
	
	hipSetDevice(dev);
	for(int i=0; i<numberOfStreamsPerDevice; i++){
		streamResult[i] = hipStreamCreate(&stream[i]);
	}
		
	//cout<<"created stream for device:"<<dev<<endl;	
}

struct AsynCalculateArgs{
	int* numberOfFeaturesPerStream;
	char** label0SamplesArray_stream_feature;
	int numOfLabel0Samples;
	char** label1SamplesArray_stream_feature;
	int numOfLabel1Samples;	
	bool** featureMasksArray_stream_feature;
	double** score;
	int device;
	hipStream_t* stream;
	GpuAcceleratedProcessor* processor;	
	int numberOfStreamsPerDevice;
	string* errorMessage;
	bool* success;
};

void asynCalculate(void* arg){
	AsynCalculateArgs* calculateArgs = (AsynCalculateArgs*) arg;	
	
	calculateArgs->processor->calculateOnStream(
		calculateArgs->numberOfFeaturesPerStream,
		calculateArgs->label0SamplesArray_stream_feature,
		calculateArgs->numOfLabel0Samples,
		calculateArgs->label1SamplesArray_stream_feature,
		calculateArgs->numOfLabel1Samples,		
		calculateArgs->featureMasksArray_stream_feature,
		calculateArgs->score,
		calculateArgs->device,
		calculateArgs->stream,
		calculateArgs->success,
		calculateArgs->errorMessage
	);
	for(int i=0; i<calculateArgs->numberOfStreamsPerDevice; i++){
		hipError_t streamResult = hipStreamDestroy(calculateArgs->stream[i]);	
	}	
	hipSetDevice(calculateArgs->device);
	hipDeviceReset();
}


Result* GpuAcceleratedProcessor::calculateOnDevice(int numOfFeatures, 
	char*** label0SamplesArray_device_stream_feature, int numOfLabel0Samples,
	char*** label1SamplesArray_device_stream_feature, int numOfLabel1Samples, 
	int** numberOfFeaturesPerStream,
	bool*** featureMasksArray_device_stream_feature,
	bool** successPerDevice, string** errorMessagePerDevice){
		
	/*
	Step 1:
		for each device
			new thread for create stream
	
	Step 2:
		create results array
	
	Step 3:
		wait for all threads join 
	
	Step 4:
		for each device	
			new thread for asyn calculation
				pass data, devId, stream for asyn calculate
				pass resultArray for passing result back
	
	Step 5:
		wait for all threads join
	
	Step 6:
		return results;
	*/
	
	Timer totalProcessing("Total Processing");
	totalProcessing.start();
	
	//get device and thread numbers	
	int deviceCount = getNumberOfDevice();	
	//ThreadPool tp(deviceCount);
	ThreadPool tp(deviceCount);
	
	int ret = tp.initialize_threadpool();
	if (ret == -1) {
		cerr << "Failed to initialize thread pool!" << endl;
		exit(EXIT_FAILURE);
	}
	
	int streamCount = getNumberOfStreamsPerDevice();
	hipStream_t stream[deviceCount][streamCount];
	hipError_t streamResult[deviceCount][streamCount];
		
	for(int dev=0; dev<deviceCount; dev++) {
		
		CreateStreamArgs* createStreamArgs = new CreateStreamArgs;		
		createStreamArgs->stream = stream[dev];
		createStreamArgs->streamResult = streamResult[dev];
		createStreamArgs->dev = dev;
		createStreamArgs->numberOfStreamsPerDevice = getNumberOfStreamsPerDevice();
		
		if(threadPoolEnabled){
			Task* t = new Task(&createStream, (void*) createStreamArgs);
			tp.add_task(t);
		}else{
			createStream(createStreamArgs);
		}
	}
	
	//do sth else when waiting...
	//get feature num 
	int numberOfDevices = getNumberOfDevice();
	int featuresPerDevice = getFeaturesPerArray(numOfFeatures, numberOfDevices);
	int numberOfStreams = getNumberOfStreamsPerDevice();
	int featuresPerStream = getFeaturesPerArray(featuresPerDevice, numberOfStreams);
	
	double ***score = (double***)malloc(numberOfDevices * sizeof(double**));
	for(int i=0; i<numberOfDevices; i++) {
		score[i] = (double**)malloc(numberOfStreams * sizeof(double*));		
		for(int j=0; j<numberOfStreams; j++){
			score[i][j] = (double*)malloc(featuresPerStream * sizeof(double));		
		}
	}
	if(isDebugEnabled()){	
		cout<<"wait for streams ready"<<endl;
	}
	if(threadPoolEnabled){
		tp.waitAll();
	}
	
	Timer processing("Processing Time");
	//note that creating stream first can save some time, can further investigate if needed
	processing.start();
	
	for(int dev=0; dev<deviceCount; dev++) {
		//cout << "[GpuAcceleratedProcessor]label0SamplesArray_device_stream_feature["<<dev<<"][0][0]=" << 0+label0SamplesArray_device_stream_feature[dev][0][0] << endl;
		AsynCalculateArgs* calculateArgs = new AsynCalculateArgs;
		calculateArgs->numberOfFeaturesPerStream = numberOfFeaturesPerStream[dev];
		calculateArgs->label0SamplesArray_stream_feature = label0SamplesArray_device_stream_feature[dev];
		calculateArgs->numOfLabel0Samples = numOfLabel0Samples;
		calculateArgs->label1SamplesArray_stream_feature = label1SamplesArray_device_stream_feature[dev];
		calculateArgs->numOfLabel1Samples = numOfLabel1Samples;
		calculateArgs->featureMasksArray_stream_feature = featureMasksArray_device_stream_feature[dev];
		calculateArgs->score = score[dev];
		calculateArgs->device = dev;
		calculateArgs->stream = stream[dev];
		calculateArgs->processor = this;
		calculateArgs->numberOfStreamsPerDevice = streamCount;
		calculateArgs->success = successPerDevice[dev];
		calculateArgs->errorMessage = errorMessagePerDevice[dev];
		
		//cout << streamResult[dev] <<endl;
		if(threadPoolEnabled){
			Task* t = new Task(&asynCalculate, (void*) calculateArgs);
			tp.add_task(t);
		}else{
			asynCalculate(calculateArgs);
		}
	}
	if(threadPoolEnabled){
		tp.waitAll();		
	}
	tp.destroy_threadpool();
	
	Result* calResult = new Result;
	calResult->scores = new double[numOfFeatures];
	
	for(int i=0;i<numOfFeatures;i++){
		int dev = i / featuresPerDevice;
		int devRemainder = i % featuresPerDevice;
		int streamId = devRemainder / featuresPerStream;
		int featureId = devRemainder % featuresPerStream;
		if(featureMasksArray_device_stream_feature[dev][streamId][featureId] != true){
			calResult->scores[i] = FEATURE_MASKED;
		}else{			
			calResult->scores[i] = score[dev][streamId][featureId];
		}
		//cout<<dev<<","<<featureId<<","<<i<<":"<<score[dev][streamId][featureId]<<endl;
	}	
		
	for(int i=0; i<numberOfDevices; i++) {		
		for(int j=0; j<numberOfStreams; j++){
			free(score[i][j]);
		}
		free(score[i]);
	}
	free(score);
	
	processing.stop();
	totalProcessing.stop();	
	calResult->startTime=processing.getStartTime();
	calResult->endTime=processing.getStopTime();	
	calResult->success=true;
	
	stringstream ss;
	
	for(int i=0; i<numberOfDevices; i++) {
		if (successPerDevice[i] == false){
			calResult->success = false;
			ss << "Device" << i << ": " << errorMessagePerDevice[i]<<"\n";
			if(isDebugEnabled()){
				cout<<"Device" << i << ": " << errorMessagePerDevice[i]<<endl;
			}
		}
	}
	if(!calResult->success){
		calResult->errorMessage = ss.str();
	}
	return calResult;	
			
	/*
		don't forget try stream features in warp size in child classes
	*/
}

Result* GpuAcceleratedProcessor::calculate(int numOfSamples, int numOfFeatures, char* sampleFeatureMatrix, bool* featureMask, char* labels){
		
	Timer pre("Pre-processing");
	pre.start();
	
	//group samples by label
	int numOfLabel0Samples = 0;
	int numOfLabel1Samples = 0;
	
	for(int j=0; j<numOfSamples; j++)
	{			
		if((int)labels[j]==0){
			numOfLabel0Samples+=1;		
		}else if((int)labels[j]==1){
			numOfLabel1Samples+=1;
		}
	}
		
	//number of array
	//device for GPU
	int numberOfDevices = getNumberOfDevice();
	int featuresPerDevice = getFeaturesPerArray(numOfFeatures, numberOfDevices);
	int numberOfStreams = getNumberOfStreamsPerDevice();
	int featuresPerStream = getFeaturesPerArray(featuresPerDevice, numberOfStreams);
			
	char ***label0SamplesArray_device_stream_feature = (char***)malloc(numberOfDevices * sizeof(char**));
	char ***label1SamplesArray_device_stream_feature = (char***)malloc(numberOfDevices * sizeof(char**));
	bool ***featureMasksArray_device_stream_feature = (bool***)malloc(numberOfDevices * sizeof(bool**));	
	int **numberOfFeaturesPerStream = (int**)malloc(numberOfDevices * sizeof(int*));	
	bool **successPerDevice = (bool**)malloc(numberOfDevices * sizeof(bool*));	
	string **errorMessagePerDevice = (string**)malloc(numberOfDevices * sizeof(string*));
	
	for(int i=0; i<numberOfDevices; i++){
		label0SamplesArray_device_stream_feature[i] = (char**)malloc(featuresPerDevice * sizeof(char*));
		label1SamplesArray_device_stream_feature[i] = (char**)malloc(featuresPerDevice * sizeof(char*));		
		featureMasksArray_device_stream_feature[i] = (bool**)malloc(featuresPerDevice * sizeof(bool*));				
		for(int j=0; j<numberOfStreams;j++){
			label0SamplesArray_device_stream_feature[i][j] = (char*)malloc(featuresPerStream * numOfLabel0Samples * sizeof(char));
			label1SamplesArray_device_stream_feature[i][j] = (char*)malloc(featuresPerStream * numOfLabel1Samples * sizeof(char));
			featureMasksArray_device_stream_feature[i][j] = (bool*)malloc(featuresPerStream * sizeof(bool));
		}
		numberOfFeaturesPerStream[i] = (int*)malloc(featuresPerStream*sizeof(int));
		memset(numberOfFeaturesPerStream[i], 0, sizeof numberOfFeaturesPerStream[i]);
		
		successPerDevice[i] = (bool*)malloc(numberOfDevices * sizeof(bool));
				
		errorMessagePerDevice[i] = (string*)malloc(numberOfDevices * sizeof(string));				
	}

	/*
	for(int i=0;i<numberOfDevices;i++){
		for(int j=0;j<numberOfStreams;j++){
			numberOfFeaturesPerStream[i][j] = 0;
		}
	}
	*/
	
	if(isDebugEnabled()){
		cout << "featuresPerDevice="<<featuresPerDevice<<", featuresPerStream="<<featuresPerStream<<endl;
	}
	
	for(int i=0;i<numOfFeatures;i++){
		int dev = i / featuresPerDevice;
		int devRemainder = i % featuresPerDevice;
		int streamId = devRemainder / featuresPerStream;
		int featureId = devRemainder % featuresPerStream;
		
		//cout<<"dev="<<dev<<", streamId="<<streamId<<", featureId="<<featureId<<endl;
		
		featureMasksArray_device_stream_feature[dev][streamId][featureId] = featureMask[i];
		
		if(featureMask[i] != true){			
			continue;
		}

		int label0Index=0;
		int label1Index=0;
		
		for(int j=0; j<numOfSamples; j++)
		{
			int index = j*numOfFeatures + i;			
			if(labels[j]==0){
				label0SamplesArray_device_stream_feature[dev][streamId][featureId * numOfLabel0Samples + label0Index]=sampleFeatureMatrix[index];
				label0Index+=1;
			}else if(labels[j]==1){
				label1SamplesArray_device_stream_feature[dev][streamId][featureId * numOfLabel1Samples + label1Index]=sampleFeatureMatrix[index];				
				label1Index+=1;
			}			
		}	
		numberOfFeaturesPerStream[dev][streamId]+=1;
	}
	
	/*
	for(int dev=0; dev<numberOfDevices; dev++) {
		cout << "[GpuAcceleratedProcessor]label0SamplesArray_device_stream_feature["<<dev<<"][0][0]=" << 0+label0SamplesArray_device_stream_feature[dev][0][0] <<", sampleFeatureMatrix[0]="<<0+sampleFeatureMatrix[0]<< endl;
	}
	*/
		
	Result* result = calculateOnDevice(numOfFeatures, 
		label0SamplesArray_device_stream_feature, numOfLabel0Samples,
		label1SamplesArray_device_stream_feature, numOfLabel1Samples, 
		numberOfFeaturesPerStream,
		featureMasksArray_device_stream_feature,
		successPerDevice, errorMessagePerDevice);

	/*
	for(int i=0; i<numOfFeatures;i++){			
		cout<<"final"<<i<<":"<<result->scores[i]<<endl;
	}	
	*/
		
	//free memory
	for(int i=0; i<numberOfDevices; i++) {
		for(int j=0; j<numberOfStreams; j++){
			free(label0SamplesArray_device_stream_feature[i][j]);
			free(label1SamplesArray_device_stream_feature[i][j]);
			free(featureMasksArray_device_stream_feature[i][j]);				
		}
		free(label0SamplesArray_device_stream_feature[i]);
		free(label1SamplesArray_device_stream_feature[i]);	
		free(featureMasksArray_device_stream_feature[i]);	
		free(numberOfFeaturesPerStream[i]);	
		free(successPerDevice[i]);
		free(errorMessagePerDevice[i]);
	}
	
	
	free(label0SamplesArray_device_stream_feature);
	free(label1SamplesArray_device_stream_feature);
	free(featureMasksArray_device_stream_feature);	
	free(numberOfFeaturesPerStream);
	free(successPerDevice);
	free(errorMessagePerDevice);
	
	return result;
}

