#include "GpuAcceleratedProcessor.h"
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

GpuAcceleratedProcessor::GpuAcceleratedProcessor(){
	this->numberOfThreadsPerBlock = 0;
	this->numberOfDevice = 0;
	this->activated = false;	
}

void GpuAcceleratedProcessor::setNumberOfThreadsPerBlock(int numberOfThreadsPerBlock)
{
		this->numberOfThreadsPerBlock = numberOfThreadsPerBlock;
}

int GpuAcceleratedProcessor::getNumberOfThreadsPerBlock()
{
		if (this->numberOfThreadsPerBlock >0){
			return numberOfThreadsPerBlock;
		}else{
			return 1024;
		}
}

void GpuAcceleratedProcessor::setNumberOfDevice(int numberOfDevice){
	this->numberOfDevice = numberOfDevice;
}

int GpuAcceleratedProcessor::getNumberOfDevice(){
	
	if (this->numberOfDevice >0){
			return numberOfDevice;
		}else{
			int deviceCount = 0;
			hipGetDeviceCount(&deviceCount);
			return deviceCount;
		}
}

int GpuAcceleratedProcessor::getNumberOfFeatureSizeTimesSampleSize2dArrays(int numOfFeatures){
	return this->getNumberOfDevice();
}

Result* GpuAcceleratedProcessor::calculate(int numOfSamples, int numOfFeatures, char* sampleTimesFeature, bool* featureMask, char* labels){
	return new Result;
}

Result* GpuAcceleratedProcessor::calculate(int numOfFeatures, 
		char** label0ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel0Samples,
		char** label1ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel1Samples, 
		bool* featureMask){
		
		//can do device muti-threading here
			
			
	return new Result;
	
}

