#include <stdio.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

int main(int argc, char* argv[]) {
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	std::cout << "number of GPU:" << deviceCount << std::endl;
}