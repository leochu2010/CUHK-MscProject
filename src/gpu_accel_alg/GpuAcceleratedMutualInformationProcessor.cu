#include "hip/hip_runtime.h"
#include "GpuAcceleratedMutualInformationProcessor.h"

#include <iostream>
#include <stdio.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

GpuAcceleratedMutualInformationProcessor::GpuAcceleratedMutualInformationProcessor(){
	parallelizationType = PARALLELIZE_ON_FEATURES;
}

__global__ void calculateMutualInformation(
	char *d_firstVector, char *d_secondVector, 
	int vectorLength, int vectorLengthPerThread,
	bool *d_featureMask,
	size_t numOfFeatures,	
	double *d_score, int device,
	int* d_excpetion)
{
	
	//gridDim.x = gridDim.y as it is 2d 
	//idx is feature index of a block on this device
	int idx = gridDim.x * blockIdx.x + blockIdx.y;
	
	/*
	if(threadIdx.x == 0){
		printf("idx=%d, blockDim.x=%d, threadIdx.x=%d",idx, blockDim.x, threadIdx.x);
	}
	*/
	
	if (idx >= numOfFeatures){		
		/*
		if(threadIdx.x == 0){
			printf("device[%d]:idx=%d >= %d \n",device,idx,numOfFeatures);
		}		
		*/
		return;
	}
	
	if (d_featureMask[idx] != true){			
		/*
		if(threadIdx.x == 0){
			printf("device[%d]:idx=%d, d_featureMask=false\n",device,idx);
		}
		*/
		return;
	}
	
	__shared__ float mutualInformation;	
	
	__shared__ int firstNumStates;
	__shared__ int secondNumStates;
	__shared__ int jointNumStates;	
	
	__shared__ int firstVectorMinVal, firstVectorMaxVal;
	__shared__ int secondVectorMinVal, secondVectorMaxVal;
	
	if(threadIdx.x == 0){
		mutualInformation= 0.0;
		firstVectorMinVal = d_firstVector[vectorLength * idx];
		firstVectorMaxVal = d_firstVector[vectorLength * idx];
		secondVectorMinVal = d_secondVector[vectorLength * idx];
		secondVectorMaxVal = d_secondVector[vectorLength * idx];
	}
	
	__syncthreads();
	
	if(vectorLengthPerThread*(threadIdx.x) < vectorLength){
		for(int i=vectorLengthPerThread*(threadIdx.x); i<vectorLengthPerThread*(threadIdx.x+1) && i<vectorLength; i++){
			int firstVectorCurrentValue = (int)(d_firstVector[vectorLength * idx + i]);
			atomicMin(&firstVectorMinVal, firstVectorCurrentValue);
			atomicMax(&firstVectorMaxVal, firstVectorCurrentValue);
			d_firstVector[vectorLength * idx + i] = firstVectorCurrentValue;
			
			int secondVectorCurrentValue = (int)(d_secondVector[vectorLength * idx + i]);
			atomicMin(&secondVectorMinVal, secondVectorCurrentValue);
			atomicMax(&secondVectorMaxVal, secondVectorCurrentValue);
			d_secondVector[vectorLength * idx + i] = secondVectorCurrentValue;
		}		
	}
	
	__syncthreads();
	if(threadIdx.x == 0){
		firstNumStates = (firstVectorMaxVal - firstVectorMinVal) + 1;
		secondNumStates = (secondVectorMaxVal - secondVectorMinVal) + 1;
		jointNumStates = firstNumStates * secondNumStates;
		if(firstNumStates > 5){
			d_excpetion[0] = -1;
		}
		if(secondNumStates > 5){
			d_excpetion[0] = -2;
		}
		if(jointNumStates > 25){
			d_excpetion[0] = -3;
		}
				
		/*
		if(idx==0){
			printf("device[%d]:idx=%d, d_excpetion = %d < 0, firstNumStates=%d, secondNumStates=%d, jointNumStates=%d \n",device,idx,d_excpetion[0],firstNumStates,secondNumStates,jointNumStates);
		}
		*/
	}
	
	if(vectorLengthPerThread*(threadIdx.x) < vectorLength){
		for(int i=vectorLengthPerThread*(threadIdx.x); i<vectorLengthPerThread*(threadIdx.x+1) && i<vectorLength; i++){    
			d_firstVector[vectorLength * idx + i] = d_firstVector[vectorLength * idx + i] - firstVectorMinVal;
			d_secondVector[vectorLength * idx + i] = d_secondVector[vectorLength * idx + i] - secondVectorMinVal;
		}
	}
	
	__syncthreads();
	
	if(d_excpetion[0] < 0){
		/*
		if(threadIdx.x == 0){
			printf("device[%d]:idx=%d, d_excpetion = %d < 0, firstNumStates=%d, secondNumStates=%d, jointNumStates=%d \n",device,idx,d_excpetion[0],firstNumStates,secondNumStates,jointNumStates);
		}
		*/
		return;
	}	
		
	__shared__ int firstStateCounts[5];
	__shared__ int secondStateCounts[5];
	__shared__ int jointStateCounts[25];
	
	__shared__ double firstStateProbs[5];
	__shared__ double secondStateProbs[5];
	__shared__ double jointStateProbs[25];
	
	if(threadIdx.x < 5){
		firstStateCounts[threadIdx.x] = 0;
		secondStateCounts[threadIdx.x] = 0;		
	
		firstStateProbs[threadIdx.x] = 0.0;
		secondStateProbs[threadIdx.x] = 0.0;
	}
	
	if(threadIdx.x < 25){
		jointStateCounts[threadIdx.x] = 0;
		jointStateProbs[threadIdx.x] = 0.0;
	}
			
	__syncthreads();
	
	/* Optimised for number of FP operations now O(states) instead of O(vectorLength) */
	if(vectorLengthPerThread*(threadIdx.x) < vectorLength){		
		for(int i=vectorLengthPerThread*(threadIdx.x); i<vectorLengthPerThread*(threadIdx.x+1) && i<vectorLength; i++){
			atomicAdd(&firstStateCounts[d_firstVector[vectorLength * idx + i]], 1);
			atomicAdd(&secondStateCounts[d_secondVector[vectorLength * idx + i]], 1);
			atomicAdd(&jointStateCounts[d_secondVector[vectorLength * idx + i] * firstNumStates + d_firstVector[vectorLength * idx + i]], 1);
		}		
	}	
	
	__shared__ int firstNumStatesPerThread;
	__shared__ int secondNumStatesPerThread;
	__shared__ int jointNumStatesPerThread;
	
	//blockDim: number of threads in a block
	if (threadIdx.x == 0){
		firstNumStatesPerThread = ceil((float)firstNumStates / blockDim.x);
		secondNumStatesPerThread = ceil((float)secondNumStates / blockDim.x);
		jointNumStatesPerThread = ceil((float)jointNumStates / blockDim.x);
	}
	
	__syncthreads();
	
	if(firstNumStatesPerThread*(threadIdx.x) < firstNumStates){
		double length = vectorLength;
		for(int i=firstNumStatesPerThread*(threadIdx.x); i<firstNumStatesPerThread*(threadIdx.x+1) && i<firstNumStates; i++){
			firstStateProbs[i] = firstStateCounts[i] / length;
		}
	}
	
	if(secondNumStatesPerThread*(threadIdx.x) < secondNumStates){
		double length = vectorLength;
		for(int i=secondNumStatesPerThread*(threadIdx.x); i<secondNumStatesPerThread*(threadIdx.x+1) && i<secondNumStates; i++){
			secondStateProbs[i] = secondStateCounts[i] / length;
		}
	}
	
	if(jointNumStatesPerThread*(threadIdx.x) < jointNumStates){
		double length = vectorLength;
		for(int i=jointNumStatesPerThread*(threadIdx.x); i<jointNumStatesPerThread*(threadIdx.x+1) && i<jointNumStates; i++){
			jointStateProbs[i] = jointStateCounts[i] / length;
		}
	}
	
	/*
	** I(X;Y) = sum sum p(xy) * log (p(xy)/p(x)p(y))
	*/	
	if(jointNumStatesPerThread*(threadIdx.x) < jointNumStates){
		for(int i=jointNumStatesPerThread*(threadIdx.x); i<jointNumStatesPerThread*(threadIdx.x+1) && i<jointNumStates; i++){
			int firstIndex = i % firstNumStates;
			int secondIndex = i / firstNumStates;
			if ((jointStateProbs[i] > 0) && (firstStateProbs[firstIndex] > 0) && (secondStateProbs[secondIndex] > 0))
			{
			  /*double division is probably more stable than multiplying two small numbers together
			  ** mutualInformation += jointStateProbs[i] * log(jointStateProbs[i] / (firstStateProbs[firstIndex] * secondStateProbs[secondIndex]));
			  */
			  double addMutualInformation = jointStateProbs[i] * log(jointStateProbs[i] / firstStateProbs[firstIndex] / secondStateProbs[secondIndex]);
			  atomicAdd(&mutualInformation, addMutualInformation);
			}
		}
	}	
	//mutualInformation /= log(2.0);	
	
	__syncthreads();
	
	if (threadIdx.x == 0){
		d_score[idx] = mutualInformation;		
		/*
		printf("device[%d]:d_score[%d]=%f\n",device,idx,mutualInformation);
		*/
	}
}

void GpuAcceleratedMutualInformationProcessor::calculateOnStream(int* numberOfFeaturesPerStream,
	char** label0SamplesArray_stream_feature, int numOfLabel0Samples,
	char** label1SamplesArray_stream_feature, int numOfLabel1Samples,
	bool** featureMasksArray_stream_feature,
	double** score,
	int device,
	hipStream_t* streams,
	bool* success, string* errorMessage){
			
	*success = true;
			
	int streamCount = getNumberOfStreamsPerDevice();
			
	if (numOfLabel0Samples != numOfLabel1Samples){
		for(int i=0; i<streamCount; i++){
			for(int j=0; j<numberOfFeaturesPerStream[i];j++){
				score[i][j]=INVALID_FEATURE;
			}
		}
		*success = false;
		*errorMessage = "numbers of label 0 and 1 samples are not the same";
		if(this->isDebugEnabled()){
			cout << "return: numbers of label 0 and 1 samples are not the same"<<endl;
		}
		return;
	}
			
	/*
	if(device == 1){
		cout<<"[GpuAcceleratedPValueProcessor]"<<"numberOfFeaturesPerStream[0]="<<numberOfFeaturesPerStream[0]<<", label0SamplesArray_stream_feature[0][0]="<<0+label0SamplesArray_stream_feature[0][0]<<endl;	
	}
	*/
	
			
	int threadSize = getNumberOfThreadsPerBlock();
	size_t samplesPerThread = ceil((numOfLabel0Samples/(float)(threadSize)));	
	
	//copy data from main memory to GPU		
	char *d_label0Array[streamCount];
	char *d_label1Array[streamCount];
	double *d_score[streamCount];
	bool *d_featureMask[streamCount];
	int *d_exception[streamCount];
		
	if(this->isDebugEnabled()){
		cout << "copy to GPU"<<endl;
	}
		
	hipSetDevice(device);
	
	int maxFeaturesPerStream = numberOfFeaturesPerStream[0];
	
	int **exception = (int**)malloc(streamCount * sizeof(int*));	

	for(int i=0; i<streamCount; i++){		
		hipMalloc(&d_label0Array[i],maxFeaturesPerStream*numOfLabel0Samples*sizeof(char));
		hipMalloc(&d_label1Array[i],maxFeaturesPerStream*numOfLabel1Samples*sizeof(char));
		hipMalloc(&d_score[i],maxFeaturesPerStream*sizeof(double));
		hipMalloc(&d_featureMask[i],maxFeaturesPerStream*sizeof(bool));
		hipMalloc(&d_exception[i],sizeof(int));		
		
		exception[i] = (int*)malloc(sizeof(int));		
		exception[i][0] = 0;
	}	
	
	for(int i=0; i<streamCount; i++){
		int features = numberOfFeaturesPerStream[i];
		hipMemcpyAsync(d_label0Array[i],label0SamplesArray_stream_feature[i],features*numOfLabel0Samples*sizeof(char),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_label1Array[i],label1SamplesArray_stream_feature[i],features*numOfLabel1Samples*sizeof(char),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_score[i],score[i],features*sizeof(double),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_featureMask[i],featureMasksArray_stream_feature[i],features*sizeof(bool),hipMemcpyHostToDevice,streams[i]);		
		hipMemcpyAsync(d_exception[i],exception[i],sizeof(int),hipMemcpyHostToDevice,streams[i]);
	}
		
	int grid2d = (int)ceil(pow(maxFeaturesPerStream,1/2.));	
	if(this->isDebugEnabled()){
		cout<<"maxFeaturesPerStream="<<maxFeaturesPerStream<<",grid2d="<<grid2d<<",numOfLabel0Samples="<<numOfLabel0Samples<<",numOfLabel1Samples="<<numOfLabel1Samples<<",threadSize="<<threadSize<<endl;
	}
	
	dim3 gridSize(grid2d,grid2d);
	
	//calculate	
	for(int i=0; i<streamCount; i++){
		calculateMutualInformation<<<gridSize, threadSize, 0, streams[i]>>>(
				d_label0Array[i], d_label1Array[i], 
				numOfLabel0Samples, samplesPerThread,
				d_featureMask[i],
				numberOfFeaturesPerStream[i],
				d_score[i], device, d_exception[i]);
	}
			
	if(this->isDebugEnabled()){
		cout<<"hipPeekAtLastError:"<<hipPeekAtLastError()<<endl;
	}	
		
	//copy result from GPU to main memory
	for(int i=0; i<streamCount; i++){
		int features = numberOfFeaturesPerStream[i];
		hipMemcpyAsync(score[i], d_score[i], features*sizeof(double), hipMemcpyDeviceToHost,streams[i]);
		hipMemcpyAsync(exception[i], d_exception[i], features*sizeof(int), hipMemcpyDeviceToHost,streams[i]);		
	}
	if(this->isDebugEnabled()){		
		cout<<"copied data from device to host"<<endl;
	}

	hipDeviceSynchronize();
		
	for(int i=0; i<streamCount; i++){
		
		if(exception[i][0] < 0){
			*success = false;
			*errorMessage = "firstNumStates/secondNumStates/jointNumStates is too big";
			if(this->isDebugEnabled()){
				cout<<"exception[i]"<<exception[i][0]<<endl;
				cout<<"error found"<<endl;
			}
		}				
	}
		
	//free cuda memory	
	//destroy streams
	for(int i=0; i<streamCount; i++){
		if(isDebugEnabled()){
			cout<<"cudafree resources"<<endl;
		}		
		hipFree(d_label1Array[i]);
		hipFree(d_label0Array[i]);
		hipFree(d_score[i]);
		hipFree(d_featureMask[i]);		
		hipFree(d_exception[i]);		
		free(exception[i]);
	}	
	free(exception);
}	