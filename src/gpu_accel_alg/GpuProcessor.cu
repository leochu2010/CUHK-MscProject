#include "hip/hip_runtime.h"
#include "GpuProcessor.h"
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

GpuProcessor::GpuProcessor(){
	this->numberOfThreadsPerBlock = 0;
	this->numberOfDevice = 0;
	this->activated = false;	
}

void GpuProcessor::setNumberOfThreadsPerBlock(int numberOfThreadsPerBlock)
{
		this->numberOfThreadsPerBlock = numberOfThreadsPerBlock;
}

int GpuProcessor::getNumberOfThreadsPerBlock()
{
		if (this->numberOfThreadsPerBlock >0){
			return numberOfThreadsPerBlock;
		}else{
			return 1024;
		}
}

void GpuProcessor::setNumberOfDevice(int numberOfDevice){
	this->numberOfDevice = numberOfDevice;
}

int GpuProcessor::getNumberOfDevice(){
	
	if (this->numberOfDevice >0){
			return numberOfDevice;
		}else{
			int deviceCount = 0;
			hipGetDeviceCount(&deviceCount);
			return deviceCount;
		}
}

int GpuProcessor::getNumberOfFeatureSizeTimesSampleSize2dArrays(int numOfFeatures){
	return this->getNumberOfDevice();
}

Result* GpuProcessor::calculate(int numOfSamples, int numOfFeatures, char* sampleTimesFeature, bool* featureMask, char* labels){
	return new Result;
}

Result* GpuProcessor::calculate(int numOfFeatures, 
		char** label0ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel0Samples,
		char** label1ProcessingUnitFeatureSizeTimesSampleSize2dArray, int numOfLabel1Samples, 
		bool* featureMask){
			
	return new Result;
	
}

__global__ void VectorAdd(int *a, int *b, int *c, int n, int threadsPerBlock)
{
	int i = blockIdx.x * threadsPerBlock + threadIdx.x;
	//printf("blockIdx.x=%d, threadsPerBlock=%d, threadIdx.x=%d, i=%d \n", blockIdx.x, threadsPerBlock, threadIdx.x, i);

	if (i < n){
		c[i] = a[i] + b[i];
	}

}
