#include "hip/hip_runtime.h"
#include "GpuAcceleratedPValueProcessor.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

using namespace std;

GpuAcceleratedPValueProcessor::GpuAcceleratedPValueProcessor(){
	parallelizationType = PARALLELIZE_ON_FEATURES;
}

//declare local function
__global__ void calculate_Pvalue(
	char *d_array1, size_t array1_size, size_t array1_size_per_thread,
	char *d_array2, size_t array2_size, size_t array2_size_per_thread, 
	double *d_score,
	bool *d_featureMask,
	size_t numOfFeatures,
	size_t NLoopPerThread,
	int device);

void GpuAcceleratedPValueProcessor::calculateOnStream(int* numberOfFeaturesPerStream,
	char** label0SamplesArray_stream_feature, int numOfLabel0Samples,
	char** label1SamplesArray_stream_feature, int numOfLabel1Samples,
	bool** featureMasksArray_stream_feature,
	double** score,
	int device,
	hipStream_t* streams,
	bool* success, string* errorMessage){
	
	*success = true;
	
	/*
	if(device == 1){
		cout<<"[GpuAcceleratedPValueProcessor]"<<"numberOfFeaturesPerStream[0]="<<numberOfFeaturesPerStream[0]<<", label0SamplesArray_stream_feature[0][0]="<<0+label0SamplesArray_stream_feature[0][0]<<endl;	
	}
	*/
	
			
	int threadSize = getNumberOfThreadsPerBlock();
	size_t label0SizePerThread = ceil((numOfLabel0Samples/(float)(threadSize)));
	size_t label1SizePerThread = ceil((numOfLabel1Samples/(float)(threadSize)));
	
	//copy data from main memory to GPU	
	int streamCount = getNumberOfStreamsPerDevice();
	char *d_label0Array[streamCount];
	char *d_label1Array[streamCount];
	double *d_score[streamCount];
	bool *d_featureMask[streamCount];
	
	if(this->isDebugEnabled()){
		cout << "copy to GPU"<<endl;
	}
		
	hipSetDevice(device);
	
	int maxFeaturesPerStream = numberOfFeaturesPerStream[0];

	for(int i=0; i<streamCount; i++){
		hipMalloc(&d_label0Array[i],maxFeaturesPerStream*numOfLabel0Samples*sizeof(char));
		hipMalloc(&d_label1Array[i],maxFeaturesPerStream*numOfLabel1Samples*sizeof(char));
		hipMalloc(&d_score[i],maxFeaturesPerStream*sizeof(double));
		hipMalloc(&d_featureMask[i],maxFeaturesPerStream*sizeof(bool));
	}	
	
	for(int i=0; i<streamCount; i++){
		int features = numberOfFeaturesPerStream[i];
		hipMemcpyAsync(d_label0Array[i],label0SamplesArray_stream_feature[i],features*numOfLabel0Samples*sizeof(char),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_label1Array[i],label1SamplesArray_stream_feature[i],features*numOfLabel1Samples*sizeof(char),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_score[i],score[i],features*sizeof(double),hipMemcpyHostToDevice,streams[i]);
		hipMemcpyAsync(d_featureMask[i],featureMasksArray_stream_feature[i],features*sizeof(bool),hipMemcpyHostToDevice,streams[i]);
	}
	
	const size_t N = 65535;
	size_t NLoopPerThread = ceil(((float)N)/threadSize);
		
	int grid2d = (int)ceil(pow(maxFeaturesPerStream,1/2.));	
	if(this->isDebugEnabled()){
		cout<<"maxFeaturesPerStream="<<maxFeaturesPerStream<<",grid2d="<<grid2d<<",numOfLabel0Samples="<<numOfLabel0Samples<<",numOfLabel1Samples="<<numOfLabel1Samples<<endl;
	}
	
	dim3 gridSize(grid2d,grid2d);
	
	//calculate	
	for(int i=0; i<streamCount; i++){
		calculate_Pvalue<<<gridSize, threadSize, 0, streams[i]>>>(
				d_label1Array[i], numOfLabel1Samples, label1SizePerThread, 
				d_label0Array[i], numOfLabel0Samples, label0SizePerThread, 
				d_score[i], 
				d_featureMask[i],
				numberOfFeaturesPerStream[i],
				NLoopPerThread,
				device);
	}
			
	if(this->isDebugEnabled()){
		cout<<"hipPeekAtLastError:"<<hipPeekAtLastError()<<endl;
	}	
		
	//copy result from GPU to main memory
	for(int i=0; i<streamCount; i++){
		int features = numberOfFeaturesPerStream[i];
		hipMemcpyAsync(score[i], d_score[i], features*sizeof(double), hipMemcpyDeviceToHost,streams[i]);
	}

	hipDeviceSynchronize();
	
	//free cuda memory	
	//destroy streams
	for(int i=0; i<streamCount; i++){
		if(isDebugEnabled()){
			cout<<"cudafree resources"<<endl;
		}	
		hipFree(d_label1Array[i]);
		hipFree(d_label0Array[i]);
		hipFree(d_score[i]);
		hipFree(d_featureMask[i]);		
	}	
}	

__global__ void calculate_Pvalue(
	char *d_array1, size_t array1_size, size_t array1_size_per_thread,
	char *d_array2, size_t array2_size, size_t array2_size_per_thread, 
	double *d_score,	
	bool *d_featureMask,
	size_t numOfFeatures,
	size_t NLoopPerThread,
	int device) {
		
	//gridDim.x = gridDim.y as it is 2d 
	//idx is feature index of a block on this device
	int idx = gridDim.x * blockIdx.x + blockIdx.y;	
	
	__shared__ float mean1;
	__shared__ float mean2;
	__shared__ float variance1;
	__shared__ float variance2;
		
	if (d_featureMask[idx] != true){			
		return;
	}
		
	if(threadIdx.x == 0){		
		mean1=0;
		mean2=0;
		variance1=0;
		variance2=0;
	}
	
	__syncthreads();
	
	if (idx < numOfFeatures){
		
		//printf("idx=%d, pitch0=%d, pitch1=%d \n", idx, pitch0, pitch1);			
				
		if(threadIdx.x == 0){
			if (array1_size <= 1) {
				d_score[idx] = 1.0;			
				return;
			}
			
			if (array2_size <= 1) {
				d_score[idx] = 1.0;
				return;
			}
		}
		
		if(array1_size_per_thread*(threadIdx.x)< array1_size){
			int m1=0;
			for(int i=array1_size_per_thread*(threadIdx.x); i<array1_size_per_thread*(threadIdx.x+1) && i<array1_size; i++){				
				m1+=d_array1[array1_size * idx + i];
				
			/*
				if(idx==0 && dev==0){
					printf("i1=%d, value=%d\n",i,array1[i]);
				}
			*/
							
			}
			
			/*
			if(idx==0){
				printf("\n threadIdx.x=%d, m1=%d",threadIdx.x, m1);
			}
			*/
			atomicAdd(&mean1,m1);
		}
					
		if(array2_size_per_thread*(threadIdx.x)< array2_size){
			int m2=0;
			for(int i=array2_size_per_thread*(threadIdx.x); i<array2_size_per_thread*(threadIdx.x+1) && i<array2_size; i++){
				m2+=d_array2[array2_size * idx + i];
				
				/*
				if(idx==1 && dev==0){
					printf("i2=%d, value=%d\n",i,d_array2[array2_size * idx + i]);
				}
				*/
				
			}
			atomicAdd(&mean2,m2);
		}
		
		__syncthreads();
		/*
		if(threadIdx.x == 0 && idx==0){
			printf("\n mean1=%f\n",mean1);
		}
		*/
		
		if(threadIdx.x == 0){
			if (mean1 == mean2) {			
				d_score[idx] = 1.0;
				return;
			}

			mean1 /= array1_size;
			mean2 /= array2_size;				
		}
				
		__syncthreads();
		
		/*
		if(threadIdx.x == 0 && idx==0 && device == 1){
			printf("\n device=%d, mean1=%f, mean2=%f",device ,mean1, mean2);			
		}
		*/
		
		if(array1_size_per_thread*(threadIdx.x) < array1_size){
			float v1 = 0;
			float v1s = 0;
			for(int i=array1_size_per_thread*(threadIdx.x); i<array1_size_per_thread*(threadIdx.x +1) && i<array1_size; i++){
				v1=(mean1-d_array1[array1_size * idx + i]);
				v1s += v1*v1; 
			}			
			atomicAdd(&variance1, v1s);
		}			

		if(array2_size_per_thread*(threadIdx.x) < array2_size){
			float v2 = 0;
			float v2s = 0;
			for(int i=array2_size_per_thread*(threadIdx.x); i<array2_size_per_thread*(threadIdx.x+1) && i<array2_size; i++){
				v2=(mean2-d_array2[array2_size * idx + i]);
				v2s += v2*v2;
			}	
			atomicAdd(&variance2, v2s);
		}		
				
		__syncthreads();
		if (threadIdx.x == 0){
			if ((variance1 == 0.0) && (variance2 == 0.0)) {
				d_score[idx] = 1.0;
				return;		
			}				
		}
		
		if(threadIdx.x == 0){
			variance1 = variance1/(array1_size-1);
			variance2 = variance2/(array2_size-1);
		}
		__syncthreads();
		
		
		/*
		if(threadIdx.x == 0 && idx==0 && device == 1){
			printf("\n device=%d, variance1=%f, variance2=%f",device ,variance1, variance2);			
		}
		*/		
		
		__shared__ float sum1;
		__shared__ float sum2;
		__shared__ double h;
		__shared__ double a;
		__shared__ double x;
		
		if (threadIdx.x == 0){
			
			const double WELCH_T_STATISTIC = (mean1-mean2)/sqrt(variance1/array1_size+variance2/array2_size);
			const double DEGREES_OF_FREEDOM = pow((double)(variance1/array1_size+variance2/array2_size),2.0)//numerator
			 /
			(
				(variance1*variance1)/(array1_size*array1_size*(array1_size-1))+
				(variance2*variance2)/(array2_size*array2_size*(array2_size-1))
			);

			a = DEGREES_OF_FREEDOM/2, x = DEGREES_OF_FREEDOM/(WELCH_T_STATISTIC*WELCH_T_STATISTIC+DEGREES_OF_FREEDOM);			
			h = x/65535;
			sum1=0;
			sum2=0;
		}
		__syncthreads();
		
		
		float s1=0;
		float s2=0;
		for(unsigned int i = (threadIdx.x)*(NLoopPerThread); i < (threadIdx.x+1)*(NLoopPerThread); i++) {
			if(i<65535){					
				s1 += (pow(h * i + h / 2.0,a-1))/(sqrt(1-(h * i + h / 2.0)));
				s2 += (pow(h * i,a-1))/(sqrt(1-h * i));
			}
		}
		atomicAdd(&sum1, s1);
		atomicAdd(&sum2, s2);		

		__syncthreads();
		
		/*
		if(device == 1 && threadIdx.x == 0 && idx==0){
			printf("device=%d, idx=%d: sum1=%f, sum2=%f, NLoopPerThread=%d\n",device, idx,sum1,sum2,NLoopPerThread);			
		}
		*/		
		
		if (threadIdx.x == 0){
			double return_value = ((h / 6.0) * ((pow(x,a-1))/(sqrt(1-x)) + 4.0 * sum1 + 2.0 * sum2))/(exp(lgamma(a)+0.57236494292470009-lgamma(a+0.5)));			
			if ((isfinite(return_value) == 0) || (return_value > 1.0)) {
				d_score[idx] = 1.0;		
			} else {							
				d_score[idx] = return_value;
				
			/*
			if ((idx==1 || idx==0)&& device == 1){
				printf("idx=%d: sum1=%f, sum2=%f\n",idx,sum1,sum2);

				printf("idx=%d: T-Test=%f, score=%f\n",idx,((mean1-mean2)/sqrt(variance1/array1_size+variance2/array2_size)),return_value);
			
				printf("idx=%d: mean1=%f,mean2=%f,variance1=%f,array1_size=%d,variance2=%f,array2_size=%d, T-Test=%f\n",idx,mean1,mean2,variance1,array1_size,variance2,array2_size,((mean1-mean2)/sqrt(variance1/array1_size+variance2/array2_size)),return_value);
			}
			*/		
				
			}
		}
		
	}	
}
